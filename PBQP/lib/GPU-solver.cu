#include "hip/hip_runtime.h"
#include "GPU-solver.cu.h"

#include <thrust/device_vector.h>
#include <thrust/extrema.h>

namespace PBQP {

namespace {

__device__
void __fillChoices(device::Graph &Graph, unsigned char *Choices, unsigned GlobalId) {
  // We assume that task with node size more than 255 is too hard
  constexpr auto MaxNodeSize = 255;
  auto NumOfNodes = Graph.getAdjMatrix().h();
  auto &AdjMatrix = Graph.getAdjMatrix();
  for (unsigned NodeIdx = 0; NodeIdx < NumOfNodes; ++NodeIdx) {
    auto CostIdx = AdjMatrix[NodeIdx][NodeIdx];
    auto &CostVect = Graph.getCostMatrix(CostIdx);
    assert(CostVect.w() == 1);
    assert(CostVect.h() <= MaxNodeSize);
    Choices[NodeIdx] = GlobalId % CostVect.h();
    GlobalId /= CostVect.h();
  }
}

__device__
Graph::Cost_t __calcMatrixesCost(device::Graph &Graph, unsigned LhsIdx, 
                                 unsigned char *Choices) {
  auto &AdjMatrix = Graph.getAdjMatrix();
  auto NumOfNodes = AdjMatrix.h();
  auto Cost = Graph::Cost_t{0};
  auto LhsChoice = Choices[LhsIdx];
  for (unsigned RhsIdx = 0; RhsIdx < NumOfNodes; ++RhsIdx) {
    if (LhsIdx == RhsIdx)
      continue;
    auto AdjCostIdx = AdjMatrix[LhsIdx][RhsIdx];
    if (AdjCostIdx >= 0) {
      auto RhsChoice = Choices[RhsIdx];
      Cost += Graph.getCostMatrix(AdjCostIdx)[LhsChoice][RhsChoice];
    }
  }
  return Cost;
}

__global__
void __calcCosts(device::Graph Graph, Graph::Cost_t *AllCosts, unsigned NumOfCombinations) {
  // We assume that task on more than 32 nodes is unsolvable, 
  //  because 2^32 at least alot
  constexpr auto MaxNumberOfNodes = 32u;
  unsigned char Choices[MaxNumberOfNodes];
  auto GlobalId = blockIdx.x * blockDim.x + threadIdx.x;
  __fillChoices(Graph, Choices, GlobalId);
  auto Cost = Graph::Cost_t{0};
  auto &AdjMatrix = Graph.getAdjMatrix();
  auto NumOfNodes = AdjMatrix.h();
  for (unsigned LhsIdx = 0; LhsIdx < NumOfNodes; ++LhsIdx) {
    Cost += __calcMatrixesCost(Graph, LhsIdx, Choices);
    auto LhsVectorCostIdx = AdjMatrix[LhsIdx][LhsIdx];
    assert(LhsVectorCostIdx >= 0);
    auto LhsChoice = Choices[LhsIdx];
    Cost += Graph.getCostMatrix(LhsVectorCostIdx)[LhsChoice][0];
  }

  if (GlobalId < NumOfCombinations)
    AllCosts[GlobalId] = Cost;
}

// Class for passing device::Graph through PassManager
struct GPUGraph final : public GPUSolver::Pass::Result {
  device::Graph Graph;

  GPUGraph(const PBQP::Graph &HostGraph) : Graph{HostGraph} {}
};

struct GPUResult final : public GPUSolver::Pass::Result {
  device::Graph Graph;
  Solution Sol;

  GPUResult(device::Graph Graph, Solution Sol = Solution{}) : Graph{Graph},
                                                              Sol{std::move(Sol)} {}
};

// Pass which creates device::Graph and passes it further
struct GraphLoader final : public GPUSolver::Pass {
  Res_t run(const Graph &Graph, Res_t PrevResult) override {
    return Res_t{new GPUGraph(Graph)};
  }
};

// Pass which finds optimal solution with full search 
//  on GPU graph received from previous pass
class FullSearchImpl final : public GPUSolver::Pass {
  static constexpr size_t BlockSize = 32;

  Solution getSolutionByIndex(device::Graph &Graph, unsigned SelectedVariant) {
    auto Res = Solution{};
    for (unsigned NodeIdx = 0; NodeIdx < Graph.size(); ++NodeIdx) {
      auto NodeCostSize = Graph.getNodeCostSize(NodeIdx);
      Res.addSelection(NodeIdx, SelectedVariant % NodeCostSize);
      SelectedVariant /= NodeCostSize;
    }
    return Res;
  }

  Solution findSolutionWithMinCost(device::Graph &Graph, 
                                   thrust::device_vector<Graph::Cost_t> Costs) {
    auto MinElemIt = thrust::min_element(Costs.begin(), Costs.end());
    assert(MinElemIt != Costs.end());
    auto MinElemIdx = std::distance(Costs.begin(), MinElemIt);
    auto Solution = getSolutionByIndex(Graph, MinElemIdx);
    Solution.addFinalCost(*MinElemIt);
    return Solution;
  }

  Solution getOptimalSolution(device::Graph &Graph) {
    auto NumOfCombinations = Graph.getNumOfCostCombinations();
    thrust::device_vector<Graph::Cost_t> 
      AllCosts(NumOfCombinations, Graph::InfCost);
    dim3 ThrBlockDim{BlockSize};
    dim3 BlockGridDim{utils::ceilDiv(NumOfCombinations, ThrBlockDim.x)};
    __calcCosts<<<BlockGridDim, ThrBlockDim>>>
      (Graph, thrust::raw_pointer_cast(AllCosts.data()), NumOfCombinations);
    hipDeviceSynchronize();
    utils::checkKernelsExec();
    return findSolutionWithMinCost(Graph, std::move(AllCosts));
  }

public:
  Res_t run(const Graph &Graph, Res_t PrevResult) override {
    auto *GPUGraphPtr = dynamic_cast<GPUGraph *>(PrevResult.get());
    if (!GPUGraphPtr)
      utils::reportFatalError("Graph hasn't been loaded to GPU");
    return Res_t{new GPUResult(GPUGraphPtr->Graph, 
                               getOptimalSolution(GPUGraphPtr->Graph))};
  }
};

// Final pass which frees GPU memory
struct GraphDeleter final : public GPUSolver::FinalPass {
  Solution getSolution(const Graph &Graph, Res_t PrevResult) override {
    auto *GPURes = dynamic_cast<GPUResult *>(PrevResult.get());
    if (!GPURes)
      utils::reportFatalError("There is no GPU solvers in PM");
    GPURes->Graph.free();
    return std::move(GPURes->Sol);
  }
};

struct FinalMock final : public GPUSolver::FinalPass {
  Solution getSolution(const Graph &Graph, Res_t PrevResult) override {
    return Solution{};
  }
};

class LoopCondition : public GPUSolver::Pass::Result {
protected:  
  bool Condition = false;

public:
  bool getCondition() const { return Condition; }
};

struct LoopConditionHandler final : public GPUSolver::Condition {
  bool check(GPUSolver::Pass::Res_t &PrevResult) override {
    auto *ResPtr = dynamic_cast<LoopCondition *>(PrevResult.get());
    if (!ResPtr)
      utils::reportFatalError("Loop header accepts only LoopCondition class");
    return ResPtr->getCondition();
  }
};

class LoopCounter : public LoopCondition {
protected:
  size_t NumOfIterations;
  size_t CurIteration = 0;

  void checkCondition() {
    Condition = CurIteration < NumOfIterations;
  }

public:
  LoopCounter(size_t NumOfIterations) : NumOfIterations{NumOfIterations} {
    checkCondition();
  } 

  void inc() {
    CurIteration++;
    checkCondition();
  }
};

struct Counter final : public GPUSolver::Pass {
  Res_t run(const Graph &Graph, Res_t PrevResult) override {
    auto *ConterResPtr = dynamic_cast<LoopCounter *>(PrevResult.get());
    if (!ConterResPtr)
      utils::reportFatalError("Counter accepts only LoopCounter result");
    ConterResPtr->inc();
    return PrevResult;
  }
};

struct CounterInit final : public GPUSolver::Pass {
  Res_t run(const Graph &Graph, Res_t PrevResult) override {
    constexpr auto IterNum = 10;
    return Res_t{new LoopCounter(IterNum)};
  }
};

} // anonymous namespace

GPUSolver::Pass::Result::~Result() {}

void GPUSolver::PassManager::addPass(Pass_t Pass, std::string Name) {
  if (Name == "")
    Name = "Pass " + std::to_string(PassPtrToName.size());
  PassPtrToName[Pass.get()] = Name;
  Stages.emplace_back(std::move(Pass));
}

void GPUSolver::PassManager::addLoopStart(Condition_t Cond) {
  LoopHeaderIdxes.push_back(Stages.size());
  Stages.emplace_back(LoopHeader{std::move(Cond)});
}

void GPUSolver::PassManager::addLoopEnd() {
  if (LoopHeaderIdxes.empty())
    utils::reportFatalError("Loop header hasn't been specified");
  auto PrevHeader = LoopHeaderIdxes.back();
  LoopHeaderIdxes.pop_back();
  auto &HeaderStage = Stages[PrevHeader];
  assert(std::holds_alternative<LoopHeader>(HeaderStage));
  std::get<LoopHeader>(HeaderStage).EndIdx = Stages.size();
  Stages.emplace_back(LoopEnd{PrevHeader});
}

GPUSolver::Res_t 
GPUSolver::PassManager::runPass(Pass_t &Pass, Res_t PrevRes, Graph &Graph) {
  auto Start = std::chrono::steady_clock::now();
  auto Res = Pass->run(Graph, std::move(PrevRes));
  auto End = std::chrono::steady_clock::now();
  PassPtrToDuration[Pass.get()] += utils::to_milliseconds(End - Start);
  return Res;
}

size_t GPUSolver::PassManager::getNextIdx(LoopHeader &Header, 
                                          Res_t &Res, size_t CurIdx) {
  if (Header.Cond->check(Res))
    return CurIdx + 1;
  return Header.EndIdx + 1;
}

Solution GPUSolver::PassManager::run(Graph Graph) {
  if (!LoopHeaderIdxes.empty())
    utils::reportFatalError("Theree is a loop header without end");
  auto Res = Pass::Res_t{};
  for (size_t CurStageIdx = 0; CurStageIdx < Stages.size();) {
    auto &CurStage = Stages[CurStageIdx];
    if (std::holds_alternative<Pass_t>(CurStage)) {
      Res = runPass(std::get<Pass_t>(CurStage), std::move(Res), Graph);
      CurStageIdx++;
      continue;
    }
    if (std::holds_alternative<LoopHeader>(CurStage)) {
      CurStageIdx = getNextIdx(std::get<LoopHeader>(CurStage), Res, CurStageIdx);
      continue;
    }
    if (std::holds_alternative<LoopEnd>(CurStage)) {
      auto &HeaderIdx = std::get<LoopEnd>(CurStage).HeaderIdx;
      auto &Header = Stages[HeaderIdx];
      assert(std::holds_alternative<LoopHeader>(Header)); 
      // We've jumped to the header
      CurStageIdx = getNextIdx(std::get<LoopHeader>(Header), Res, HeaderIdx);
      continue;
    }
  }

  auto *SolutionPtr = dynamic_cast<FinalSolution *>(Res.get());
  if (!SolutionPtr)
    utils::reportFatalError("Invalid final pass");
  
  return SolutionPtr->getFinalSolution(std::move(Graph));
}

GPUSolver::PassManager::Profile_t
GPUSolver::PassManager::getProfileInfo() const {
  auto Res = std::vector<std::pair<std::string, size_t>>{};
  std::transform(Stages.begin(), Stages.end(),
                 std::back_inserter(Res),
                [&](const auto &Stage) {
                  if (std::holds_alternative<LoopHeader>(Stage))
                    return std::pair<std::string, size_t>{"Loop header", 0};
                  if (std::holds_alternative<LoopEnd>(Stage))
                    return std::pair<std::string, size_t>{"Loop end", 0};
                  auto PassPtr = std::get<Pass_t>(Stage).get();
                  auto NameIt = PassPtrToName.find(PassPtr);
                  assert(NameIt != PassPtrToName.end());
                  auto DurationIt = PassPtrToDuration.find(PassPtr);
                  assert(DurationIt != PassPtrToDuration.end());
                  return std::pair<std::string, size_t>{NameIt->second, 
                                                        DurationIt->second};
                });
  return Res;
}

Solution GPUSolver::solve(Graph Task) {
  this->addPasses(PM);
  return PM.run(std::move(Task));           
}

void GPUFullSearch::addPasses(PassManager &PM) {
  PM.addPass(Pass_t{new GraphLoader});
  PM.addPass(Pass_t{new FullSearchImpl}, "GPU full search");
  PM.addPass(Pass_t{new GraphDeleter});
}

void HeuristicSolver::addPasses(PassManager &PM) {
  PM.addPass(Pass_t{new CounterInit});
  PM.addLoopStart(Condition_t{new LoopConditionHandler});
    PM.addPass(Pass_t{new Counter});
  PM.addLoopEnd();
  PM.addPass(Pass_t{new FinalMock});
}

} // namespace PBQP