
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <thread>

extern __global__ void MatMul();

__device__ int A[1];

int main() {
  std::cout << "Cuda" << std::endl;
  //int *GlobA = nullptr;                     //this is device memory. so we can't print it
  int LocA = 2;
  //cudaMalloc(&GlobA, sizeof(GlobA));

  hipMemcpyToSymbol(HIP_SYMBOL(A), &LocA, sizeof(int));
  MatMul<<<10, 1>>>();
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&LocA, HIP_SYMBOL(A), sizeof(int));

  std::cout << "A: " << LocA << std::endl;
  //cudaFree(GlobA);
  std::cout << "Cuda done"  << std::endl;
}